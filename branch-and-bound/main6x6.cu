#include "hip/hip_runtime.h"
#include <bits/stdc++.h> 
#include <iostream>
#include "instancias.cc"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
__global__
void vecAddKernel(int* costMatriz, bool *tempAllocated, bool *allocated, int minJob, int min, int imachine)
{
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(!allocated[i] && tempAllocated[i] && costMatriz[i] < min) {
        minJob = i;
        min = costMatriz[i];
    }
}

class Node {
    public:
        Node(int machine, int job, bool* allocated, int problemSize, Node* parent = nullptr) {
            this->problemSize = problemSize;
            this->machine = machine;
            this->job = job;
            this->parent = parent;
            this->allocated = new bool[problemSize];


            for(int iallocated = 0; iallocated < this->problemSize; iallocated++) {
                this->allocated[iallocated] = allocated[iallocated];
            }
            this->allocated[this->job] = true;
        }

        int problemSize;
        int machine;
        int job;
        Node* parent;
        int pathCost;
        bool* allocated;
        int cost;

        int calculateLeastCost(int costMatriz[6][6]) {
            int size = 6 * sizeof(int);
            int sizeb = 6 * sizeof(bool);
            int cost = 0;

            bool tempAllocated[this->problemSize] = {true};

            for(int imachine = this->machine+1; imachine < this->problemSize; imachine++) {
                int min = INT_MAX;
                int minJob = -1;
                /*  for(int ijob = 0; ijob < this->problemSize; ijob++) {
                    if(!allocated[ijob] && tempAllocated[ijob] && costMatriz[imachine][ijob] < min) {
                        minJob = ijob;
                        min = costMatriz[imachine][ijob];
                    }
                }  */

                int *d_A;
                bool *t_allocate, *t_allo;
                int *tminJob, *tmin, *tmachine;


                hipMalloc((void **) &d_A, size);
                hipMemcpy(d_A, costMatriz[imachine], size, hipMemcpyHostToDevice);
                hipMalloc((void **) &t_allocate, sizeb);
                hipMemcpy(t_allocate, tempAllocated, sizeb, hipMemcpyHostToDevice);
                hipMalloc((void **) &t_allo, sizeb);
                hipMemcpy(t_allo, allocated, sizeb, hipMemcpyHostToDevice);

                vecAddKernel<<<ceil(6/256.00), 256>>>(d_A, t_allocate, t_allo, minJob, min, machine);

                hipMemcpy(&minJob, tminJob, sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&min, tmin, sizeof(int), hipMemcpyDeviceToHost);

                cost += min;
                tempAllocated[minJob] = false;

                hipFree(d_A); hipFree(t_allocate); hipFree(tminJob); hipFree(tmin); hipFree(tmachine); hipFree(t_allo);

                
            }

            return cost;
        } 
};



class BranchAndBound {
    public:
    
        BranchAndBound(int costMatriz[6][6]) {
            this->problemSize = 6;
            for(int i = 0; i < 6; i++) {
                for(int e = 0; e < 6; e++) {
                    this->costMatriz[i][e] = costMatriz[i][e];
                }
            }
        }

        int minimumCostSearch() {
            std::priority_queue<Node*, std::vector<Node*>, comp> queue;

            bool allocated[this->problemSize] = {false}; 
            Node* nodeZero = new Node(-1, -1, allocated, this->problemSize);
            nodeZero->pathCost = 0;
            nodeZero->cost = 0;
            nodeZero->machine = -1;

            queue.push(nodeZero);

            while(!queue.empty()) {
                Node* nodeMinCost = queue.top();
                queue.pop();
                int machine = nodeMinCost->machine + 1;

                if(machine == this->problemSize) {
                    printAssignments(nodeMinCost);
                    return nodeMinCost->cost;
                }

                for(int job = 0; job < this->problemSize; job++) {
                    if(!nodeMinCost->allocated[job]) {
                        Node* newNode = new Node(machine, job, nodeMinCost->allocated, this->problemSize, nodeMinCost);
                        newNode->pathCost = nodeMinCost->pathCost + costMatriz[machine][job];
                        newNode->cost = newNode->pathCost + newNode->calculateLeastCost(costMatriz);
                        queue.push(newNode);
                    }
                }
            }
        }

        void printAssignments(Node *min) { 
            if(min->parent==NULL) 
                return; 

            printAssignments(min->parent); 
            std::cout << "Assign Worker " << char(min->machine + 'A') 
                << " to Job " << min->job << std::endl; 
        } 

    private:

        int problemSize;
        int costMatriz[6][6]; 

        struct comp { 
            bool operator()(const Node* lhs, 
                        const Node* rhs) const
            { 
                return lhs->cost > rhs->cost; 
            } 
        };
};

int main() {

    struct timeval ti, tf;
    double tiempo;
    gettimeofday(&ti, NULL);


    BranchAndBound babP5(p8);
    babP5.minimumCostSearch();
    

    gettimeofday(&tf, NULL);
    tiempo = (tf.tv_sec - ti.tv_sec)*1000 + (tf.tv_usec - ti.tv_usec)/1000;
    printf("time bbRun 2:       %.28f s\n", tiempo/1000);

    return EXIT_SUCCESS;
}